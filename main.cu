#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#define MAX_FILE_SIZE 1048576
#define STORAGE_SIZE 1085440
#define DATAFILE "./data.bin"
#define OUTPUTFILE "./snapshot.bin"
#define G_WRITE 1
#define G_READ 2
#define RM 3
#define RM_RF 4
#define LS_S 5
#define LS_D 6
#define dataHead 36864
#define BASE 1030

typedef unsigned char uchar;
typedef uint32_t u32;
typedef unsigned short u16;

__device__ __managed__ uchar *volume;
__device__ __managed__ uchar tempFCB[30];
__device__ __managed__ uchar temp[64];

void init_volume() {
	int i;
    memset(volume, 0, STORAGE_SIZE * sizeof(uchar));
	for(i = 0; i < 1024; i++){
		volume[i] = 1; 
	}//0 ~ 1023 : map to free space
	//1024 ~ 1025 : file count
	//1026 ~ 1029 : time stamp 
	//file format : 20byte file name + 4 byte address + 4 byte time + 2 byte size
	//total : 1030 + 30*1024 = 31750
}
int loadBinaryFile(char *fileName, uchar *vol, int fileSize) {
    FILE *fp = fopen(fileName, "rb");
    int size;
    
	fseek(fp, 0, SEEK_END);
    size = ftell(fp);
    rewind(fp);
    fread(vol, sizeof(uchar), size, fp);
	fclose(fp);
    return size;
}

void writeBinaryFile(char *fileName, uchar *output, int fileSize) {
    FILE *fp = fopen(fileName, "wb+");
    fwrite(output, sizeof(uchar), fileSize, fp);
	fclose(fp);
}
__device__ void cpy(char *dest, char const *src){
	int i = 0;
	for(i = 0; i < 20 && src[i] != '\0';i++){
		dest[i] = src[i];
	}
}
__device__ int length(char const *a){
	int i = 0;
	
	if(a == NULL) return 0;
	while(a[i] !='\0' && i < 20){
		i += 1;
	}
	return i;
}
__device__ bool cmp(char const *a, char const *b){
	int len = length(a);
	int lenb = length(b);
	int i;

	if(len == 0 || lenb == 0) return false;
	if(len != lenb) return false;
	else{
		for(i = 0; i < len; i++){
			if(a[i] != b[i]) return false;
		}
		return true;
	}
	return false;
}
__device__ u32* getAddr(int i){
	return (u32*)volume + BASE + i*30 + 20;
}
__device__ char* getName(int i){
	return (char*)volume + BASE + i*30;
}
__device__ u16 getSize(int i){
	return (u16)*(volume + BASE + i*30 + 28);
}
__device__ u32 getTime(int i){
	return (u32)*(volume + BASE + i*30 + 24);
}
__device__ void swapFCB(int a, int b){
	int i;
	int indexA = BASE + a*30;
	int indexB = BASE + b*30;
	for(i = 0; i < 26; i++){
		tempFCB[i] = volume[indexA + i];
	}
	for(i = 0; i < 26; i++){
		volume[indexA + i] = volume[indexB + i];
	}
	for(i = 0; i < 26; i++){
		volume[indexB + i] = tempFCB[i];
	}
}
__device__ void swapContent(int a, int b){
	int i, j;
	int indexA = dataHead + (a << 10);
	int indexB = dataHead + (b << 10);
	for(i = 0; i < 16; i++){
		for(j = 0; j < 64; j++){
			temp[j] = volume[indexA + (i << 6) + j];
		}
		for(j = 0; j < 64; j++){
			volume[indexA + (i << 6) + j] = volume[indexB + (i << 6) + j];
		}
		for(j = 0; j < 64; j++){
			volume[indexB + (i <<6) + j] = temp[j];
		}
	}
}
__device__ void sortBySize(){
	int i, j;
	u16 *fileCount = (u16*)volume + 1024;
	
	for(i = 0; i < *fileCount; i++){
		for(j = 0; j < *fileCount - i -1; j++){
			if(getSize(j) < getSize(j+1)){
				swapFCB(j, j+1);
				swapContent(j, j+1);
			}
			else if(getSize(j) == getSize(j+1)){
				if(getTime(j) > getTime(j+1)){
					swapFCB(j, j+1);
					swapContent(j, j+1);
				}
			}	
		}
	}
}
__device__ void sortByTime(){
	int i, j;
	u16 *fileCount = (u16*)volume + 1024;
	
	for(i = 0; i < *fileCount; i++){
		for(j = 0; j < *fileCount - i - 1; j++){
			if(getTime(j) > getTime(j+1)){
				swapFCB(j, j+1);
				swapContent(j, j+1);
			}
		}
	}
}

__device__ u32 findFreeSpace(){
	int i;
	for(i = 0; i < 1024; i++){
		if(volume[i] == 1){
			volume[i] = 0;
			break;
		}
	}
	return dataHead +  (i << 10);
}

__device__ u32 open(char const *name, int type) {
	u16 *fileCount = (u16*)(volume + 1024);
	u32 *address = NULL;
	int i;
	char *fileName;
	printf("open\n");
	
	for(i = 0; i < *fileCount; i++){//linear search for file
		fileName = (char*)&volume[BASE +  i*30];
		if(cmp(name, fileName) == true){
			address = getAddr(i);
			if(type == G_WRITE)memset(volume + *address, 0, sizeof(char) * 1024);
			break;
		}
	}
	if(address == NULL && type == G_WRITE){//file not found, create new
		cpy((char*)(volume + BASE + (*fileCount)*30), name);
		volume[*fileCount] = 0;
		*fileCount += 1;
		printf("create new file\n");
		address = getAddr(fileCount);
	}
	printf("fileCount : %d\n", *fileCount);
    return *address;
}

__device__ void remove(u32 address){
	int index = (address - dataHead) >> 10;
	//int i;
	//u32 *addr;
	u16 *fileCount = (u16*)volume + 1024;
	
	volume[index] = 1;
	memset(volume + BASE + index*30, 0, 30 * sizeof(char));//clean FCB
	if(volume[dataHead - 1] == 0)sortByTime();
	else sortBySize();
	*fileCount -= 1;
/*
	for(i = 0; i < *fileCount; i++){
		addr = volume + BASE + i*26 + 20;	
		if(*addr == address){
			memset(volume+BASE+i*26, 0, 26*sizeof(char));//clean FCB
			break;
		}
	}
*/
}

__device__ void write(uchar *src, int size, u32 fp) {
	int i;
	int index = (fp - dataHead) >> 10;
	u32 *time = (u32*)volume + 1026;
	printf("%d\n", fp);
	for(i = 0; i < size; i ++){
		volume[fp + i] = src[i];
	}
	*(volume + BASE + index*30 + 24) = *time;
	*(volume + BASE + index*30 + 28) = size;
	*time = *time + 1;
	printf("write\n");
}

__device__ void read(uchar *dest, int size, u32 fp) {
	int i;
	u32 *time = (u32*)volume + 1026;
	
	for(i = 0; i < size; i++){
		dest[i] = volume[fp + i];
	}
	*time = *time + 1;
}

__device__ void printFCB_D(){
	u16 *fileCount = (u16*)(volume + 1024);
	int i;
	
	printf("===sort by file time===\n");
	for(i = 0; i < *fileCount; i++){
		printf("%s\n", getName(i));
	}
}
__device__ void printFCB_S(){
	u16 *fileCount = (u16*)(volume + 1024);
	int i;

	printf("===sort by file size===\n");
	for(i = 0; i < *fileCount; i++){
		printf("%s %d\n", getName(i), getSize(i));
	}
}
__device__ void gsys(int cmd) {
	printf("gsys\n");
	if(cmd == LS_S){
		if(volume[dataHead - 1] == 0)sortBySize();
		printFCB_S();
	}
	else if(cmd == LS_D){
		if(volume[dataHead - 1] == 1)sortByTime();
		printFCB_D();
	}
}
__device__ void gsys(int cmd, char const *fileName) {
	u16* fileCount = (u16*)volume + 1024;
	int i;
	if(cmd == RM){
		for(i = 0; i < *fileCount; i++){
			if(cmp(getName(i), fileName) == true){
				remove(*(getAddr(i)));
				break;
			}
		}	
	}
}
__global__ void mykernel(uchar *input, uchar *output) {
    printf("kernel start\n");
	//####kernel start####
    u32 fp = open("t.txt\0", G_WRITE);
    printf("fp: %d\n");
	write(input, 64, fp);
    fp = open("b.txt\0", G_WRITE);
    write(input+32, 32, fp);
    fp = open("t.txt\0", G_WRITE);
    write(input+32, 32, fp);
    read(output, 32, fp);
    gsys(LS_D);
    gsys(LS_S);
    fp = open("b.txt\0", G_WRITE);
    write(input + 64, 12, fp);
    gsys(LS_S);
    gsys(LS_D);
    gsys(RM, "t.txt\0");
    gsys(LS_S);
    //####kernel end####
}

int main() {
    hipMallocManaged(&volume, STORAGE_SIZE);
    init_volume();

    uchar *input, *output;
    hipMallocManaged(&input, MAX_FILE_SIZE);
    hipMallocManaged(&output, MAX_FILE_SIZE);
    loadBinaryFile(DATAFILE, input, MAX_FILE_SIZE);

    hipSetDevice(1);
    mykernel<<<1, 1>>>(input, output);
    hipDeviceSynchronize();
    writeBinaryFile(OUTPUTFILE, output, MAX_FILE_SIZE);
    hipDeviceReset();

    return 0;
}
